#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "load_ckpt.h"
#include "cnpy.h"

// Thread block size
#define BLOCK_SIZE 32

// CUDA kernel for matrix multiplication (optimized with shared memory)
__global__ void matrixMulKernel(float* A, float* B, float* C, int M, int K, int N) {
    // Block index
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Thread index
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;

    // Shared memory for tile of input matrices
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Each thread computes one element of C
    float sum = 0.0f;

    // Loop over all tiles
    int numTiles = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for (int t = 0; t < numTiles; t++) {
        // Load one tile of A and B into shared memory
        int aRow = blockRow * BLOCK_SIZE + threadRow;
        int aCol = t * BLOCK_SIZE + threadCol;
        int bRow = t * BLOCK_SIZE + threadRow;
        int bCol = blockCol * BLOCK_SIZE + threadCol;

        // Boundary check
        if (aRow < M && aCol < K)
            As[threadRow][threadCol] = A[aRow * K + aCol];
        else
            As[threadRow][threadCol] = 0.0f;

        if (bRow < K && bCol < N)
            Bs[threadRow][threadCol] = B[bRow * N + bCol];
        else
            Bs[threadRow][threadCol] = 0.0f;

        __syncthreads(); // Wait for all threads to load data

        // Calculate partial dot product
        for (int k = 0; k < BLOCK_SIZE; k++) {
            sum += As[threadRow][k] * Bs[k][threadCol];
        }

        __syncthreads(); // Wait for all threads to finish using the tile
    }

    // Write result
    int cRow = blockRow * BLOCK_SIZE + threadRow;
    int cCol = blockCol * BLOCK_SIZE + threadCol;
    if (cRow < M && cCol < N) {
        C[cRow * N + cCol] = sum;
    }
}
// CUDA Error checking
// #define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
// void check(hipError_t result, char const *const func, const char *const file, int const line) {
//     if (result) {
//         printf("CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
//                static_cast<unsigned int>(result), hipGetErrorString(result), func);
//         exit(EXIT_FAILURE);
//     }
// }

int main(int argc, char* argv[]) {
    // Check if correct number of arguments are provided
 /*   if (argc != 5) {
        printf("Usage: %s <M> <K> <N> <V>\n", argv[0]);
        printf("Where:\n");
        printf("  M = number of rows in matrix A\n");
        printf("  K = number of columns in matrix A / rows in matrix B\n");
        printf("  N = number of columns in matrix B / rows in matrix X\n");
        printf("  V = number of columns in matrix X\n");
        return 1;
    }*/

    srand(time(NULL));

    float *h_A = nullptr;
    float *h_B = nullptr;
    float *h_X = nullptr;

    std::string A_dir = "./lora/gemma-2-2b-loraA.npy";
    std::string B_dir = "./lora/gemma-2-2b-loraB.npy";
    std::string X_dir = "./inputs/x_1024.npy";

    auto [A_rows, A_cols] =  load_ckpt_float(A_dir, h_A);
    auto [B_rows,B_cols] = load_ckpt_float(B_dir,h_B);
    auto [X_rows,X_cols] = load_ckpt_float(X_dir,h_X);

     size_t bytes_A = A_rows * A_cols * sizeof(float);
    size_t bytes_B = B_rows * B_cols * sizeof(float);
    size_t bytes_W = A_rows * B_cols * sizeof(float);
    size_t bytes_X = X_rows * X_cols * sizeof(float);
    size_t bytes_Y = A_rows * X_cols * sizeof(float);

    float *h_W = (float*)malloc(bytes_W);
	float *h_Y = (float*)malloc(bytes_Y);

    // Allocate device memory
    float *d_A, *d_B, *d_W, *d_X, *d_Y;
    hipMalloc(&d_A, bytes_A);
    hipMalloc(&d_B, bytes_B);
    hipMalloc(&d_W, bytes_W);
    hipMalloc(&d_X, bytes_X);
    hipMalloc(&d_Y, bytes_Y);

    // Copy host to device
    hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes_B, hipMemcpyHostToDevice);
    hipMemcpy(d_X, h_X, bytes_X, hipMemcpyHostToDevice);

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid1((B_cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (A_rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dimGrid2((X_cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (A_rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    printf("\nRunning Calculation: Y = (A.B).X\n");
    printf("----------------------------------\n");

    // Start total timing
    hipEventRecord(start);

    // Step 1: W = A.B
    hipEventRecord(start);
    matrixMulKernel<<<dimGrid1, dimBlock>>>(d_A, d_B, d_W, A_rows, A_cols, B_cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for W = A.B: %f ms\n", milliseconds);
    float time_W = milliseconds;

    // Step 2: Y = W.X
    hipEventRecord(start);
    matrixMulKernel<<<dimGrid2, dimBlock>>>(d_W, d_X, d_Y, A_rows, B_cols, X_cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for Y = W.X: %f ms\n", milliseconds);
    float time_Y = milliseconds;

    // End total timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total time for Y = (A.B).X: %f ms\n", time_W + time_Y);

    // Copy result back to host
    hipMemcpy(h_Y, d_Y, bytes_Y, hipMemcpyDeviceToHost);

    // Print first and last elements of results
    printf("\nResults:\n");
    printf("Y[0][0] = %f (First element)\n", h_Y[0]);
    //printf("Y[%d][%d] = %f (Last element)\n", A_rows-1, X_cols-1, h_Y[(A_rows*X_cols)-1]);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_W);
    hipFree(d_X);
    hipFree(d_Y);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(h_A);
    free(h_B);
    free(h_W);
    free(h_X);
    free(h_Y);

    return 0;
}
