// ==========================================
// main.cu
// ==========================================
#include <iostream>
#include <vector>
#include <string>
#include "hadamard.h"
#include "matrix_ops.h"
#include "utils.h"
#include <hip/hip_runtime.h>

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <x_input_filename.bin>\n";
        return 1;
    }

    std::string xfile = argv[1];

    // Create CUDA events for profiling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Step 1: Load C values and locations
    std::vector<float> C_values;
    std::vector<int> C_locs;
    int C_rows, C_cols;
    load_sparse_C("inputs_bin/C_values.bin", "inputs_bin/C_locs.bin", C_values, C_locs, C_rows, C_cols);

    // Step 2: Find next powers of 2
    int next_pow2_rows = next_power_of_2(C_rows);
    int next_pow2_cols = next_power_of_2(C_cols);

    // Step 3: Create Hadamard matrices
    float* d_H_row = nullptr;
    float* d_H_col = nullptr;
    create_hadamard_matrix(next_pow2_rows, &d_H_row);
    create_hadamard_matrix(next_pow2_cols, &d_H_col);

    // Start timing
    hipEventRecord(start);

    // Step 4: Reconstruct and pad C matrix
    float* d_C = nullptr;
    reconstruct_C(C_values, C_locs, C_rows, C_cols, next_pow2_rows, next_pow2_cols, &d_C);
    
    // Ensure C is fully ready before using it
    hipDeviceSynchronize();

    // Step 5: Calculate deltaW = H * C * H
    float* d_deltaW = nullptr;
    calculate_deltaW(d_H_row, d_C, d_H_col, next_pow2_rows, next_pow2_cols, &d_deltaW);

    // Ensure deltaW is fully ready
    hipDeviceSynchronize();

    // Step 6: Compute Y = deltaW * X
    compute_Y(d_deltaW, xfile, next_pow2_cols, next_pow2_rows);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "\n All computations done on CUDA!\n";
    std::cout << " CUDA Execution Time: " << milliseconds << " ms\n";

    // Cleanup
    hipFree(d_H_row);
    hipFree(d_H_col);
    hipFree(d_C);
    hipFree(d_deltaW);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
