#include <iostream>
#include "load_ckpt.h"
#include "matmul.cuh"
#include "construct_frame.h"


int main(int argc, char* argv[]) {
    int ct_mat_rows = atoi(argv[1]);  // 2304;
    int ct_mat_cols = atoi(argv[2]);  //1024;
    int l_m = atoi(argv[3]);  // 2;
    int l_n = atoi(argv[4]);  // 2;
    int k_m = ct_mat_rows / l_m;
    int k_n = ct_mat_cols / l_n;
    int num_tokens = atoi(argv[5]);  // 16;
    int threads_per_block = atoi(argv[6]);  // 256;
    std::string ct_directory = argv[7];  // "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/value/frame/Gemma-2-2b-frame-value-CT.npy";
    std::string locs_directory = argv[8];  // "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/value/frame/Gemma-2-2b-frame-value-locs.npy";
    std::string x_directory = argv[9];  // "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/inputs/x_16.npy";

    // load the coefficients
    float* ct = nullptr; 
    auto [ct_rows, ct_cols] = load_ckpt_float(ct_directory, ct);

    // load the locations
    int* locs = nullptr; 
    auto [locs_rows, locs_cols] = load_ckpt_int(locs_directory, locs);

    // load the tokens
    float* x = nullptr; 
    auto [x_rows, x_cols] = load_ckpt_float(x_directory, x);

    float* tff_m = nullptr;
    float* tff_n = nullptr;

    tff_m = construct_real_tff(k_m, l_m/2, ct_mat_rows/2);
    tff_n = construct_real_tff(k_n, l_n/2, ct_mat_cols/2);

    // multiply tff_m * ct_mat * tff_n.T in cuda
    float *d_tff_m, *d_tff_n, *d_ct, *d_D1, *d_y, *d_x;
    float *d_tff_m_unfold, *d_tff_n_unfold;
    int *d_locs;

    // Allocate memory
    hipMalloc(&d_tff_m, ct_mat_rows * ct_mat_rows * sizeof(float));
    hipMalloc(&d_tff_n, ct_mat_cols * ct_mat_cols * sizeof(float));
    hipMalloc(&d_tff_m_unfold, ct_mat_rows * ct_cols * sizeof(float));
    hipMalloc(&d_tff_n_unfold, ct_cols * ct_mat_cols * sizeof(float));

    hipMalloc(&d_ct, ct_cols * sizeof(float));
    hipMalloc(&d_locs, locs_rows * locs_cols * sizeof(int));

    hipMalloc(&d_y, num_tokens * ct_mat_cols * sizeof(float));
    hipMalloc(&d_x, num_tokens * ct_mat_rows * sizeof(float));

    hipMalloc(&d_D1, num_tokens * ct_cols * sizeof(float));

    // Copy data
    hipMemcpy(d_tff_m, tff_m, ct_mat_rows * ct_mat_rows * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tff_n, tff_n, ct_mat_cols * ct_mat_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ct, ct, ct_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_locs, locs, locs_rows * locs_cols * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, num_tokens * ct_mat_rows * sizeof(float), hipMemcpyHostToDevice);

    // Timing with CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    frame_compute_y_3(  d_tff_m,
                        d_tff_n,
                        d_ct,
                        d_locs,
                        d_tff_m_unfold,
                        d_tff_n_unfold,
                        d_x,
                        d_D1,
                        d_y,
                        num_tokens,
                        ct_mat_rows,
                        ct_mat_cols,
                        ct_cols,
                        threads_per_block);
    hipEventRecord(stop);

    // Wait for kernel to finish
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // // Copy result back to host
    // int bytes = num_tokens * ct_mat_cols * sizeof(float);
    // float* h_y = new float[num_tokens * ct_mat_cols];
    // hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost);

    std::cout << milliseconds << std::endl;

    // for DEBUG
    // save_array("/home/harsha/proj/ece759-final-proj/temp.npy", ct_mat, ct_mat_rows * ct_mat_cols);

    delete[] ct;
    delete[] locs;
    delete[] tff_m;
    delete[] tff_n;
    // delete[] h_y;
    hipFree(d_tff_m);
    hipFree(d_tff_n);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}