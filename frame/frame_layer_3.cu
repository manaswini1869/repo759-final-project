#include <iostream>
#include "load_ckpt.h"
#include "matmul.cuh"
#include "construct_frame.h"


int main() {
    int ct_mat_rows = 2304, ct_mat_cols = 1024;
    int k_m = 1152, l_m = 2, k_n = 512, l_n = 2;
    int num_tokens = 16;
    int threads_per_block = 256;

    // load the coefficients
    std::string ct_directory = "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/value/frame/Gemma-2-2b-frame-value-CT.npy";
    float* ct = nullptr; 
    auto [ct_rows, ct_cols] = load_ckpt_float(ct_directory, ct);

    // load the locations
    std::string locs_directory = "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/value/frame/Gemma-2-2b-frame-value-locs.npy";
    int* locs = nullptr; 
    auto [locs_rows, locs_cols] = load_ckpt_int(locs_directory, locs);

    // load the tokens
    std::string x_directory = "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/inputs/x_16.npy";
    float* x = nullptr; 
    auto [x_rows, x_cols] = load_ckpt_float(x_directory, x);

    float* tff_m = nullptr;
    float* tff_n = nullptr;

    tff_m = construct_real_tff(k_m, l_m/2, ct_mat_rows/2);
    tff_n = construct_real_tff(k_n, l_n/2, ct_mat_cols/2);

    // multiply tff_m * ct_mat * tff_n.T in cuda
    float *d_tff_m, *d_tff_n, *d_ct, *d_D1, *d_y, *d_x;
    float *d_tff_m_unfold, *d_tff_n_unfold;
    int *d_locs;

    // Allocate memory
    hipMalloc(&d_tff_m, ct_mat_rows * ct_mat_rows * sizeof(float));
    hipMalloc(&d_tff_n, ct_mat_cols * ct_mat_cols * sizeof(float));
    hipMalloc(&d_tff_m_unfold, ct_mat_rows * ct_cols * sizeof(float));
    hipMalloc(&d_tff_n_unfold, ct_cols * ct_mat_cols * sizeof(float));

    hipMalloc(&d_ct, ct_cols * sizeof(float));
    hipMalloc(&d_locs, locs_rows * locs_cols * sizeof(int));

    hipMalloc(&d_y, num_tokens * ct_mat_cols * sizeof(float));
    hipMalloc(&d_x, num_tokens * ct_mat_rows * sizeof(float));

    hipMalloc(&d_D1, num_tokens * ct_cols * sizeof(float));

    // Copy data
    hipMemcpy(d_tff_m, tff_m, ct_mat_rows * ct_mat_rows * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tff_n, tff_n, ct_mat_cols * ct_mat_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ct, ct, ct_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_locs, locs, locs_rows * locs_cols * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, num_tokens * ct_mat_rows * sizeof(float), hipMemcpyHostToDevice);

    // Timing with CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    frame_compute_y_3(  d_tff_m,
                        d_tff_n,
                        d_ct,
                        d_locs,
                        d_tff_m_unfold,
                        d_tff_n_unfold,
                        d_x,
                        d_D1,
                        d_y,
                        num_tokens,
                        ct_mat_rows,
                        ct_mat_cols,
                        ct_cols,
                        threads_per_block);
    hipEventRecord(stop);

    // Wait for kernel to finish
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    int bytes = num_tokens * ct_mat_cols * sizeof(float);
    float* h_y = new float[num_tokens * ct_mat_cols];
    hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost);

    // print first 10 elements of h_y
    for (int i = 0; i < 10; ++i) {
        std::cout << h_y[i] << " ";
    }
    // Print the last element
    std::cout << h_y[num_tokens * ct_mat_cols - 1] << std::endl;
    std::cout << milliseconds << std::endl;

    // for DEBUG
    // save_array("/home/harsha/proj/ece759-final-proj/temp.npy", ct_mat, ct_mat_rows * ct_mat_cols);

    delete[] ct;
    delete[] locs;
    delete[] tff_m;
    delete[] tff_n;
    delete[] h_y;
    hipFree(d_tff_m);
    hipFree(d_tff_n);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}