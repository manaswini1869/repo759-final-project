#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <cstddef>
#include <cstdio>


// Does C = A * B
__global__ void frame_matmul_kernel(const float* A, const float* B, float* C, size_t m, size_t k, size_t n)
{
    // Calculate the row and column index for the element to compute
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = idx / n; // Row index
    size_t col = idx % n; // Column index

    // Check if the indices are within bounds
    if (row < m && col < n)
    {
        float sum = 0.0f;
        for (size_t i = 0; i < k; ++i)
        {
            sum += A[row * k + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

// Does C = A^T * B
__global__ void frame_matmul_transpose_kernel(const float* A, const float* B, float* C, size_t m, size_t k, size_t n)
{
    // Calculate the row and column index for the element to compute
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = idx / n; // Row index
    size_t col = idx % n; // Column index

    // Check if the indices are within bounds
    if (row < m && col < n)
    {
        float sum = 0.0f;
        for (size_t i = 0; i < k; ++i)
        {
            sum += A[i * m + row] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}


// Does C = block(A) * B
__global__ void frame_block_matmul_kernel(const float* b_A, const float* B, float* C, size_t k_m, size_t k_n, size_t l_m, size_t l_n, size_t rows, size_t cols)
{
    // Calculate the row and column index for the element to compute
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = idx / cols; // Row index
    size_t col = idx % cols; // Column index

    size_t num_blocks = (k_m < k_n) ? k_m : k_n;

    // Check if the indices are within bounds
    if (row < rows && col < cols)
    {
        float sum = 0.0f;

        int block_id = row / l_m;
        if (block_id < num_blocks)
        {
            int block_row = row % l_m;
            for (size_t i = 0; i < l_n; ++i)
            {
                sum += b_A[block_id * l_m * l_n + block_row * l_n + i] * B[(block_id * l_m + i) * cols + col];
            }
        }
        C[row * cols + col] = sum;
    }
}


void frame_compute_dw(const float* tff_m, const float* tff_n, const float* ct_mat, float* D, float* result, size_t m, size_t n, unsigned int threads_per_block)
{
    size_t num_blocks = (m * n + threads_per_block - 1) / threads_per_block;

    // Launch the kernel
    frame_matmul_transpose_kernel<<<num_blocks, threads_per_block>>>(tff_m, ct_mat, D, m, m, n);

    // launch the kernel for the right multiplication
    frame_matmul_kernel<<<num_blocks, threads_per_block>>>(D, tff_n, result, m, n, n);
}


void frame_compute_y(const float* x, const float* dw, float* y, size_t n, size_t m, size_t d, unsigned int threads_per_block)
{
    size_t num_blocks = (n * d + threads_per_block - 1) / threads_per_block;

    // Launch the kernel
    frame_matmul_kernel<<<num_blocks, threads_per_block>>>(x, dw, y, n, m, d);
}




void frame_compute_y_2( const float* tff_m,
                        const float* tff_n,
                        const float* bct,
                        const float* x,
                        float* D1,
                        float* D2,
                        float* y,
                        size_t num_tokens,
                        size_t ct_mat_rows,
                        size_t ct_mat_cols,
                        size_t k_m, size_t k_n, size_t l_m, size_t l_n,
                        unsigned int threads_per_block)
{

    size_t num_blocks_d1 = (num_tokens * ct_mat_rows + threads_per_block - 1) / threads_per_block;
    frame_matmul_transpose_kernel<<<num_blocks_d1, threads_per_block>>>(x, tff_m, D1, num_tokens, ct_mat_rows, ct_mat_rows);

    size_t num_blocks_d2 = (ct_mat_rows * ct_mat_cols + threads_per_block - 1) / threads_per_block;
    frame_block_matmul_kernel<<<num_blocks_d2, threads_per_block>>>(bct, tff_n, D2, k_m, k_n, l_m, l_n, ct_mat_rows, ct_mat_cols);

    size_t num_blocks_y = (num_tokens * ct_mat_cols + threads_per_block - 1) / threads_per_block;
    frame_matmul_kernel<<<num_blocks_y, threads_per_block>>>(D1, D2, y, num_tokens, ct_mat_rows, ct_mat_cols);

}