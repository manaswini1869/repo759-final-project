#include <iostream>
#include "load_ckpt.h"
#include "matmul.cuh"
#include "construct_frame.h"


int main(int argc, char* argv[]) {
    int ct_mat_rows = atoi(argv[1]);  // 2304;
    int ct_mat_cols = atoi(argv[2]);  //1024;
    int l_m = atoi(argv[3]);  // 2;
    int l_n = atoi(argv[4]);  // 2;
    int k_m = ct_mat_rows / l_m;
    int k_n = ct_mat_cols / l_n;
    int num_tokens = atoi(argv[5]);  // 16;
    int threads_per_block = atoi(argv[6]);  // 256;
    std::string ct_directory = argv[7];  // "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/value/frame/Gemma-2-2b-frame-value-CT.npy";
    std::string locs_directory = argv[8];  // "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/value/frame/Gemma-2-2b-frame-value-locs.npy";
    std::string x_directory = argv[9];  // "/home/harsha/proj/ece759-final-proj/checkpoints/Gemma-2-2b/inputs/x_16.npy";

    // load the coefficients
    float* ct = nullptr; 
    auto [ct_rows, ct_cols] = load_ckpt_float(ct_directory, ct);

    // load the locations
    int* locs = nullptr; 
    auto [locs_rows, locs_cols] = load_ckpt_int(locs_directory, locs);

    // load the tokens
    float* x = nullptr; 
    auto [x_rows, x_cols] = load_ckpt_float(x_directory, x);

    int num_blocks = (k_m < k_n) ? k_m : k_n;
    int num_coeffs_per_block = l_m * l_n;
    int total_num_coeffs = num_blocks * num_coeffs_per_block;
    float* bct = new float[total_num_coeffs];

    // init bct to zeros
    for (size_t i = 0; i < total_num_coeffs; ++i) {
        bct[i] = 0.0f;
    }

    // copy the values of ct into the ct_mat at locations specified by locs
    for (size_t i = 0; i < locs_cols; ++i) {
        int blk_id = locs[i]/l_m;
        int row_in_blk = locs[i] % l_m;
        int col_in_blk = locs[locs_cols + i] % l_n;

        int coeff_loc = blk_id * num_coeffs_per_block + row_in_blk * l_n + col_in_blk;
        bct[coeff_loc] = ct[i];
    }

    float* tff_m = nullptr;
    float* tff_n = nullptr;

    tff_m = construct_real_tff(k_m, l_m/2, ct_mat_rows/2);
    tff_n = construct_real_tff(k_n, l_n/2, ct_mat_cols/2);

    // multiply tff_m * ct_mat * tff_n.T in cuda
    float *d_tff_m, *d_tff_n, *d_bct, *d_D1, *d_D2, *d_y, *d_x;

    // Allocate memory
    hipMalloc(&d_tff_m, ct_mat_rows * ct_mat_rows * sizeof(float));
    hipMalloc(&d_tff_n, ct_mat_cols * ct_mat_cols * sizeof(float));
    hipMalloc(&d_bct, total_num_coeffs * sizeof(float));

    hipMalloc(&d_y, num_tokens * ct_mat_cols * sizeof(float));
    hipMalloc(&d_x, num_tokens * ct_mat_rows * sizeof(float));

    hipMalloc(&d_D1, num_tokens * ct_mat_rows * sizeof(float));
    hipMalloc(&d_D2, ct_mat_rows * ct_mat_cols * sizeof(float));

    // Copy data
    hipMemcpy(d_tff_m, tff_m, ct_mat_rows * ct_mat_rows * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tff_n, tff_n, ct_mat_cols * ct_mat_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bct, bct, total_num_coeffs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, num_tokens * ct_mat_rows * sizeof(float), hipMemcpyHostToDevice);

    // Timing with CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    frame_compute_y_2(  d_tff_m,
                        d_tff_n,
                        d_bct,
                        d_x,
                        d_D1,
                        d_D2,
                        d_y,
                        num_tokens,
                        ct_mat_rows,
                        ct_mat_cols,
                        k_m, k_n, l_m, l_n,
                        threads_per_block);
    hipEventRecord(stop);

    // Wait for kernel to finish
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // // Copy result back to host
    // int bytes = num_tokens * ct_mat_cols * sizeof(float);
    // float* h_y = new float[num_tokens * ct_mat_cols];
    // hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost);

    std::cout << milliseconds << std::endl;

    // for DEBUG
    // save_array("/home/harsha/proj/ece759-final-proj/temp.npy", ct_mat, ct_mat_rows * ct_mat_cols);

    delete[] ct;
    delete[] bct;
    delete[] locs;
    delete[] tff_m;
    delete[] tff_n;
    // delete[] h_y;
    hipFree(d_tff_m);
    hipFree(d_tff_n);
    hipFree(d_y);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}