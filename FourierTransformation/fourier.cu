// nvcc fourier_finetune.cu -lcufft -o fourier_finetune
#include <iostream>
#include <iomanip>
#include <vector>
#include <random>
#include <set>
#include <hipfft/hipfft.h>


void random_init(float* matrix, int size) {
    std::mt19937 gen(42);
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (int i = 0; i < size; ++i) {
        matrix[i] = dist(gen);
    }
}

void apply_frequency_delta(hipfftComplex* freq, int rows, int cols, int C) {
    std::set<std::pair<int, int>> freq_positions;
    std::mt19937 gen(123);
    std::uniform_real_distribution<float> dist(-0.1f, 0.1f);

    while (freq_positions.size() < C) {
        int i = gen() % rows;
        int j = gen() % cols;
        freq_positions.insert({i, j});
    }

    for (const auto& pos : freq_positions) {
        int idx = pos.first * cols + pos.second;
        freq[idx].x += dist(gen);
        freq[idx].y += dist(gen);
    }
}

void print_matrix(const float* data, int rows, int cols, const char* title = "Matrix") {
    std::cout << title << ":\n";
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << std::fixed << std::setw(8) << std::setprecision(2)
                      << data[i * cols + j] << " ";
        }
        std::cout << "\n";
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <rows> <cols> <C>\n";
        return EXIT_FAILURE;
    }

    int rows, cols, C;
    *rows = std::stoi(argv[1]);
    *cols = std::stoi(argv[2]);
    *C = std::stoi(argv[3]);
    if (*rows <= 0 || *cols <= 0 || *C <= 0) {
        std::cerr << "Error: rows, cols, and C must be given and should be positive\n";
        exit(EXIT_FAILURE);
    }
    int size = rows * cols;

    // Host memory
    std::vector<float> h_W(size);
    std::vector<float> h_W_prime(size);

    random_init(h_W.data(), size);

    float* d_W;
    float* d_W_prime;
    hipfftComplex* d_freq;

    CHECK_CUDA(hipMalloc(&d_W, sizeof(float) * size));
    CHECK_CUDA(hipMalloc(&d_W_prime, sizeof(float) * size));
    CHECK_CUDA(hipMalloc(&d_freq, sizeof(hipfftComplex) * size));
    CHECK_CUDA(hipMemcpy(d_W, h_W.data(), sizeof(float) * size, hipMemcpyHostToDevice));

    hipfftHandle plan_forward, plan_inverse;
    hipfftPlan2d(&plan_forward, rows, cols, HIPFFT_R2C);
    hipfftPlan2d(&plan_inverse, rows, cols, HIPFFT_C2R);

    hipfftExecR2C(plan_forward, d_W, d_freq);
    CHECK_CUDA(hipDeviceSynchronize());

    std::vector<hipfftComplex> h_freq(size);
    CHECK_CUDA(hipMemcpy(h_freq.data(), d_freq, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost));
    apply_frequency_delta(h_freq.data(), rows, cols, C);
    CHECK_CUDA(hipMemcpy(d_freq, h_freq.data(), sizeof(hipfftComplex) * size, hipMemcpyHostToDevice));

    // Inverse FFT: F⁻¹(F(W) + ΔF) = W'
    hipfftExecC2R(plan_inverse, d_freq, d_W_prime);
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy back result
    CHECK_CUDA(hipMemcpy(h_W_prime.data(), d_W_prime, sizeof(float) * size, hipMemcpyDeviceToHost));

    for (auto& val : h_W_prime) {
        val /= size;
    }

    std::cout << "Frequency fine-tuning complete.\n";
    print_matrix(h_W_prime.data(), rows, cols, "W' (after ΔFourier)");

    // Cleanup
    hipFree(d_W);
    hipFree(d_W_prime);
    hipFree(d_freq);
    hipfftDestroy(plan_forward);
    hipfftDestroy(plan_inverse);

    return 0;
}
